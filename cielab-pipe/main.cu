#include <stdio.h>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

int main(int argc, char *argv[])
{
    // sanity check
    if (argc != 2)
    {
        printf("Usage: %s <path-to-image>\n", argv[0]);
        return EXIT_FAILURE;
    }

    // load image
    cv::Mat orig_img;
    orig_img = cv::imread(argv[1]);

    // convert image to I420
    cv::Mat orig_img_i420;
    cv::cvtColor(orig_img, orig_img_i420, cv::COLOR_BGRA2YUV_YV12);

    // prepare data pointers
    void *h_data = (void *) orig_img_i420.data, *d_data;

    // allocate device memory
    int count = orig_img_i420.total() * orig_img_i420.elemSize();
    hipMalloc(&d_data, count);

    // copy data to device
    hipMemcpy(d_data, h_data, count, hipMemcpyHostToDevice);

    // TODO: delete this, testing only
    cv::Mat proc_img_i420;
    proc_img_i420 = orig_img_i420;

    // copy data back to host
    hipMemcpy(h_data, d_data, count, hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_data);

    // save processed image
    cv::Mat proc_img;
    cv::cvtColor(proc_img_i420, proc_img, cv::COLOR_YUV2BGRA_YV12);
    if (not cv::imwrite("./cielab-pipe-processed.jpg", proc_img))
    {
        printf("Could not write processed image\n");
        return EXIT_FAILURE;
    }

    // all good
    return EXIT_SUCCESS;
}

