
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char *argv[])
{
    // sanity check
    if (argc != 2)
    {
        printf("Usage: %s <path-to-image>\n", argv[0]);
        return EXIT_FAILURE;
    }

    // all good
    return EXIT_SUCCESS;
}

