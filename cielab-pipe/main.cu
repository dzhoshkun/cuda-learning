#include <stdio.h>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <nppi.h>

int main(int argc, char *argv[])
{
    // sanity check
    if (argc != 2)
    {
        printf("Usage: %s <path-to-image>\n", argv[0]);
        return EXIT_FAILURE;
    }

    // load image
    cv::Mat orig_img;
    orig_img = cv::imread(argv[1]);
    int n_rows = orig_img.rows, n_cols = orig_img.cols;
    // assert both dimensions divisible by 4,
    // needed for I420 memory offset computations
    assert((int)(n_cols * 0.25) == (n_cols * 0.25));
    assert((int)(n_rows * 0.25) == (n_rows * 0.25));

    // convert image to I420
    cv::Mat orig_img_i420;
    cv::cvtColor(orig_img, orig_img_i420, cv::COLOR_BGRA2YUV_YV12);

    // prepare data pointers
    Npp8u *h_data = (Npp8u *) orig_img_i420.data, *d_data;

    // allocate device memory
    int count = orig_img_i420.total() * orig_img_i420.elemSize();
    hipMalloc((void **) &d_data, count);

    // copy data to device
    hipMemcpy(d_data, h_data, count, hipMemcpyHostToDevice);

    // allocate memory for BGR data on device
    Npp8u *d_data_bgr;
    hipMalloc(&d_data_bgr, 3 * n_cols * n_rows * sizeof(Npp8u));

    // convert I420 to BGR on device
    {
        Npp8u *pSrc[3], *pDst = d_data_bgr;
        pSrc[0] = (Npp8u *) d_data;
        pSrc[1] = (Npp8u *) &d_data[n_rows * n_cols]; // TODO: is address op used correctly
        // TODO: assert offset integer
        pSrc[2] = (Npp8u *) &d_data[(int)(n_rows * n_cols * 1.25)]; // TODO: is address op used correctly
        int rSrcStep[3], nDstStep = 3 * n_cols;
        rSrcStep[0] = n_cols;
        // TODO: assert offset integer
        rSrcStep[1] = (int)(n_cols * 0.25);
        // TODO: assert offset integer
        rSrcStep[2] = (int)(n_cols * 0.25);
        NppiSize oSizeROI;
        oSizeROI.width = n_cols;
        oSizeROI.height = n_rows; // TODO: check row vs. col order!
        NppStatus ret = nppiYUV420ToBGR_8u_P3C3R(pSrc, rSrcStep, pDst, nDstStep, oSizeROI);
        if (ret != 0)
            printf("I420-to-BGR returned %d\n", ret);
    }
    hipFree(d_data);

    // allocate memory for CIELAB data on device
    Npp8u *d_data_lab;
    hipMalloc(&d_data_lab, 3 * n_cols * n_rows * sizeof(Npp8u));

    // convert BGR to CIELAB on device
    {
        Npp8u *pSrc = d_data_bgr, *pDst = d_data_lab;
        int nSrcStep = 3 * n_cols, nDstStep = 3 * n_cols;
        NppiSize oSizeROI;
        oSizeROI.width = n_cols;
        oSizeROI.height = n_rows; // TODO: check row vs. col order!
        NppStatus ret = nppiBGRToLab_8u_C3R(pSrc, nSrcStep, pDst, nDstStep, oSizeROI);
        if (ret != 0)
            printf("BGR-to-CIELAB returned %d\n", ret);
    }
    hipFree(d_data_bgr);

    // allocate memory for BGR on device
    Npp8u *d_proc_data_bgr;
    hipMalloc(&d_proc_data_bgr, 3 * n_cols * n_rows * sizeof(Npp8u));

    // convert CIELAB back to BGR
    {
        Npp8u *pSrc = d_data_lab, *pDst = d_proc_data_bgr;
        int nSrcStep = 3 * n_cols, nDstStep = 3 * n_cols;
        NppiSize oSizeROI;
        oSizeROI.width = n_cols;
        oSizeROI.height = n_rows; // TODO: check row vs. col order!
        NppStatus ret = nppiLabToBGR_8u_C3R(pSrc, nSrcStep, pDst, nDstStep, oSizeROI);
        if (ret != 0)
            printf("CIELAB-to-BGR returned %d\n", ret);
    }
    hipFree(d_data_lab);

    // allocate memory for I420 on device
    Npp8u *d_proc_data;
    hipMalloc(&d_proc_data, count);

    // convert BGR back to I420
    {
        Npp8u *pSrc = d_proc_data_bgr, *pDst[3];
        pDst[0] = d_proc_data;
        pDst[1] = &d_proc_data[n_cols * n_rows]; // TODO: is address op used correctly
        // TODO: assert offset integer
        pDst[2] = &d_proc_data[(int)(n_cols * n_rows * 1.25)]; // TODO: is address op used correctly
        int nSrcStep = 3 * n_cols, nDstStep = n_cols; // TODO: why nDstStep scalar?
        NppiSize oSizeROI;
        oSizeROI.width = n_cols;
        oSizeROI.height = n_rows; // TODO: check row vs. col order!
        NppStatus ret = nppiBGRToYUV_8u_C3P3R(pSrc, nSrcStep, pDst, nDstStep, oSizeROI);
        if (ret != 0)
            printf("BGR-to-I420 returned %d\n", ret);
    }
    hipFree(d_proc_data_bgr);

    // copy data back to host
    cv::Mat proc_img_i420 = cv::Mat::zeros(orig_img_i420.size(), orig_img_i420.type());
    h_data = proc_img_i420.data;
    hipMemcpy(h_data, d_proc_data, count, hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_proc_data);

    hipDeviceSynchronize();

    // save processed image
    cv::Mat proc_img;
    cv::cvtColor(proc_img_i420, proc_img, cv::COLOR_YUV2BGRA_YV12);
    if (not cv::imwrite("./cielab-pipe-processed.jpg", proc_img))
    {
        printf("Could not write processed image\n");
        return EXIT_FAILURE;
    }

    // all good
    return EXIT_SUCCESS;
}

