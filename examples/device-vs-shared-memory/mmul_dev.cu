/* Matrix multiplication using device memory, adapted from:
 * https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}

int main(int argc, char * argv[])
{
    if (argc < 3)
    {
        printf("Usage: %s <width> <height>\n", argv[0]);
        return EXIT_FAILURE;
    }

    Matrix A, B, C;

    A.width = atoi(argv[1]);
    A.height = atoi(argv[2]);
    A.elements = (float *) malloc(A.width * A.height * sizeof(float));
    for (int i = 0; i < A.width; i++)
        for (int j = 0; j < A.height; j++)
            A.elements[j * A.width + i] = j + i;

    B.width = A.height;
    B.height = A.width;
    B.elements = (float *) malloc(B.width * B.height * sizeof(float));
    for (int i = 0; i < B.width; i++)
        for (int j = 0; j < B.height; j++)
            B.elements[j * B.width + i] = j + i;

    printf("Multiplying %dx%d matrix with %dx%d matrix\n",
           A.width, A.height, B.width, B.height);

    C.width = B.width;
    C.height = A.height;
    C.elements = (float *) malloc(C.width * C.height * sizeof(float));

    MatMul(A, B, C);

    delete [] A.elements;
    delete [] B.elements;
    delete [] C.elements;

    return EXIT_SUCCESS;
}

