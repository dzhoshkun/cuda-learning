/**
 * Simple example provided by NVIDIA for profiling and understanding GPU acceleration.
 * Source: https://devblogs.nvidia.com/even-easier-introduction-cuda/
 * Retrieved: 30 June 2018
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
// Kernel function to add the elements of two arrays

__global__
void add_1_thread(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

__global__
void add_1_block(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

__global__
void add_grid(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(int argc, char * argv[])
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = atoi(argv[2]);
  int numBlocks = atoi(argv[1]);
  int numThreads = numBlocks * blockSize;
  if (numThreads != N)
    printf("%d elements can't be processed by %d threads!\n",
           N, numThreads);
  add_grid<<<numBlocks, blockSize>>>(N, x, y);
  add_1_block<<<1, blockSize>>>(N, x, y);
  add_1_thread<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}

