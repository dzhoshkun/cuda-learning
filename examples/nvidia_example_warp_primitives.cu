
#include <hip/hip_runtime.h>
#include <stdio.h>


__device__
void warp_reduce_add()
{
    int val = threadIdx.x;
    printf("Thread %d has value %d before op\n", threadIdx.x, val);
    #define FULL_MASK 0xffffffff
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(FULL_MASK, val, offset);
    printf("Thread %d has value %d after op\n", threadIdx.x, val);
}


__global__
void add()
{
    warp_reduce_add();
}


int main(int argc, char * argv[])
{
    int nthreads = atoi(argv[1]);
    add<<<1, nthreads>>>();
    hipDeviceSynchronize();
    return 0;
}

