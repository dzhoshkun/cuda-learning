
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(float * A, float * B, float * C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

void VecPrint(float * V, int len)
{
    int to_print = 10;
    if (to_print > len)
        to_print = len;
    for (int i=0; i<to_print; i++)
    {
        printf("%4.2f", V[i]);
        if (i<to_print-1)
            printf(", ");
    }
    if (to_print < len)
        printf("...");
    printf("\n");
}

int main()
{
    int N = 1024;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A, h_B and h_C in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    for (int i=0; i<N; i++)
    {
        h_A[i] = 2*i;
        h_B[i] = 4*i;
        h_C[i] = 0;
    }

    // Print initialised vectors
    VecPrint(h_A, N);
    VecPrint(h_B, N);
    VecPrint(h_C, N);

    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Run the add kernel
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // Print result
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    VecPrint(h_C, N);    

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;

    return 0;
}
